#include "FFT.h"
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include ""

#define C(X) if((X)!=hipSuccess) return false

bool FFT(float* input, std::complex<float>* output, const unsigned long size, const unsigned long batch) {
	hipfftReal *in, *in_h;
	hipfftComplex *out, *out_h;

	if (hipMalloc((void**)&in, size * batch * sizeof(hipfftReal)) != hipSuccess)
		return false;

	if (hipHostMalloc((void**)&in_h, size * batch * sizeof(hipfftReal)) != hipSuccess) {
		hipFree(in);
		return false;
	}

	if (hipMalloc((void**)&out, (size / 2 + 1) * batch * sizeof(hipfftComplex)) != hipSuccess) {
		hipFree(in);
		hipHostFree(in_h);
		return false;
	}
	if (hipHostMalloc((void**)&out_h, (size / 2 + 1) * batch * sizeof(hipfftComplex)) != hipSuccess) {
		hipFree(in);
		hipHostFree(in_h);
		hipFree(out);
		return false;
	}

	auto p = in_h;

	for (unsigned long i = 0; i < size* batch; ++i)
		*p++ = (hipfftReal)*input++;



	if (hipMemcpy(in, in_h, size * batch * sizeof(hipfftReal), hipMemcpyHostToDevice) != hipSuccess) {
		hipFree(in);
		hipHostFree(in_h);
		hipFree(out);
		hipHostFree(out_h);
		return false;
	}

	hipfftHandle plan;
	//hipfftPlan1d(&plan, size, HIPFFT_R2C, batch);
	int n = size;
	hipfftPlanMany(&plan, 1, &n,
		NULL, 1, size,
		NULL, 1, (size / 2 + 1), HIPFFT_R2C, batch);
	hipfftExecR2C(plan, in, out);
	hipDeviceSynchronize();
	
	if (hipMemcpy(out_h, out, (size / 2 + 1) * batch * sizeof(hipfftComplex), hipMemcpyDeviceToHost) != hipSuccess) {
		hipFree(in);
		hipHostFree(in_h);
		hipFree(out);
		hipHostFree(out_h);
		return false;
	}

	hipfftDestroy(plan);
	
	for (unsigned long i = 0; i < (size / 2 + 1) * batch; ++i)
		output[i] = std::complex<float>(out_h[i].x, out_h[i].y);

	hipFree(in);
	hipHostFree(in_h);
	hipFree(out);
	hipHostFree(out_h);
	return true;
}
